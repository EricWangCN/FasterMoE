
#include <hip/hip_runtime.h>
#ifdef FMOE_USE_NCCL

#include <cstdlib>
#include <vector>
#include <torch/extension.h>
#include <c10/cuda/CUDAGuard.h>

#include "fused_compute.cuh"

long pipeline_gran = -1;

std::vector<torch::Tensor> _fused_forward(
        torch::Tensor input_buf,
        std::vector<std::vector<std::vector<torch::Tensor>>> params,
        torch::Tensor local_expert_count,
        torch::Tensor global_expert_count,
        torch::Tensor stored_models,
        torch::Tensor fwd_expert_count,
        long n_workers, bool has_bias) {

    if (pipeline_gran == -1) {
        char* p = getenv("FMOE_FUSE_GRAN");
        if (p) {
            pipeline_gran = atoi(p);
        } else {
            pipeline_gran = 4;
        }
    }

    auto smgr = getCudaStreamManager(input_buf.device().index());
    int rank;
    NCCL_SAFE_CALL(ncclCommUserRank(smgr->ncclcomm, &rank));

    const auto num_expert = local_expert_count.size(0) / n_workers;
    const auto d_hidden = params[rank][0][0].size(1);
    const auto d_model = params[rank][0][0].size(2);

    auto global_batch_size = fwd_expert_count.sum().item<int>();
    auto global_input_buf = input_buf.new_zeros({global_batch_size, d_model});
    auto global_middle_buf = input_buf.new_zeros({global_batch_size, d_hidden});
    auto cache_middle_buf = input_buf.new_zeros({global_batch_size, d_hidden});
    auto global_output_buf = input_buf.new_zeros({global_batch_size, d_model});
    auto output_buf = input_buf.new_zeros({input_buf.size(0), d_model}).add(159);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input_buf.scalar_type(), 
            "fmoe_cuda_fused_forward", ([&] {
        fmoe_cuda_fused_forward_impl(
            input_buf.data_ptr<scalar_t>(),
            params,

            global_input_buf.data_ptr<scalar_t>(),
            global_middle_buf.data_ptr<scalar_t>(),
            cache_middle_buf.data_ptr<scalar_t>(),
            global_output_buf.data_ptr<scalar_t>(),
            output_buf.data_ptr<scalar_t>(),

            local_expert_count.data_ptr<long>(),
            global_expert_count.data_ptr<long>(),
            stored_models.data_ptr<bool>(),
            d_model, d_hidden, num_expert, rank, n_workers, has_bias,
            pipeline_gran, smgr);
    }));
    return {output_buf, global_input_buf, global_middle_buf, global_output_buf};
}

std::vector<torch::Tensor> _fused_backward(
        torch::Tensor input_buf,
        std::vector<std::vector<std::vector<torch::Tensor>>> params,
        torch::Tensor middle_buf,
        torch::Tensor output_buf,
        torch::Tensor grad_out,
        torch::Tensor local_expert_count,
        torch::Tensor global_expert_count,
        torch::Tensor stored_models,
        
        long global_batch_size,
        long buf_batch_size,
        long n_workers, bool has_bias) {
    const auto num_expert = local_expert_count.size(0) / n_workers;
    
    auto smgr = getCudaStreamManager(input_buf.device().index());
    int rank;
    ncclCommUserRank(smgr->ncclcomm, &rank);
    
    const auto d_hidden = params[rank][0][0].size(1);
    const auto d_model = params[rank][0][0].size(2);


    auto global_grad_out = input_buf.new_zeros({global_batch_size, d_model});
    auto grad_middle = input_buf.new_zeros({global_batch_size, d_hidden});
    auto cache_grad_middle = input_buf.new_zeros({global_batch_size, d_hidden});
    auto global_grad_in = input_buf.new_zeros({global_batch_size, d_model});

    auto cache_middle_buf = middle_buf.clone();

    auto grad_in = input_buf.new_zeros({buf_batch_size, d_model});
    
    for (auto node : params)
        for (auto expert : node)
            for (int i = 0; i < expert.size(); i++) {
                // create the respective gradient of each tensor
                expert[i].mutable_grad() = input_buf.new_zeros(expert[i].sizes());
            }

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input_buf.scalar_type(), 
            "fmoe_cuda_fused_backward", ([&] {
        fmoe_cuda_fused_backward_impl(
            input_buf.data_ptr<scalar_t>(),
            params,

            middle_buf.data_ptr<scalar_t>(),
            cache_middle_buf.data_ptr<scalar_t>(),
            output_buf.data_ptr<scalar_t>(),
            grad_out.data_ptr<scalar_t>(),

            global_grad_out.data_ptr<scalar_t>(),
            global_grad_in.data_ptr<scalar_t>(),

            grad_middle.data_ptr<scalar_t>(),
            cache_grad_middle.data_ptr<scalar_t>(),
            grad_in.data_ptr<scalar_t>(),

            local_expert_count.data_ptr<long>(),
            global_expert_count.data_ptr<long>(),
            stored_models.data_ptr<bool>(),
            d_model, d_hidden, num_expert, rank, n_workers, has_bias,
            pipeline_gran, smgr);
    }));
    return {grad_in,};
}

#endif

